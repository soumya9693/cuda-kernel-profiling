#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(const float *A, const float *B, float *C, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) C[i] = A[i] + B[i];
}

int main() {
    int N = 1 << 16;
    size_t size = N * sizeof(float);

    // Host memory
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    for (int i = 0; i < N; i++) {
        h_A[i] = i * 0.5f;
        h_B[i] = i * 0.25f;
    }

    // Device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify result
    for (int i = 0; i < 5; i++)
        printf("%f + %f = %f\n", h_A[i], h_B[i], h_C[i]);

    // Cleanup
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(h_A); free(h_B); free(h_C);

    return 0;
}
